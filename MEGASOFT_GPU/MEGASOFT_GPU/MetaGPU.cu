#include "hip/hip_runtime.h"
#include"MetaGPU.cuh"


__device__ __host__  void gpu_logBeta(double* res, double m, double n)
{
	// Beta Function on CUDA
	// https://stackoverflow.com/questions/15158297/beta-pdf-function-for-cuda/15159945
	*res = log(exp(lgamma(m) + lgamma(n) - lgamma(n + m)));
	//*res = log(boost::math::beta(m, n));
	return;
}
__device__ __host__ void gpu_chiSquareComplemented(double* res, double v, double x) 
{
	// Incomplete Gamma Function
	// https://www.cs.rit.edu/~ark/lectures/cuda01/c2html.php?file=9
	*res = 1 - (gammp(v / 2.0, x / 2.0));
	return;
}