#pragma once
// NVIDIA CUDA LIBRARY
#include"gpu.cuh"

void printGPUInfo(int* GPU_EXISTS) {
    

    hipGetDeviceCount(GPU_EXISTS);
    for (int i = 0; i < *GPU_EXISTS; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
            prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
            prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
            2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
    }
    return;
}