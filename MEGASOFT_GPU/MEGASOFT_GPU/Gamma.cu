#include "hip/hip_runtime.h"
//******************************************************************************
//
// File:    Gamma.cu
// Author:  Alan Kaminsky
// Version: 02-Feb-2012
//
// This source file is copyright (C) 2012 by Parallel Crypto LLC. All rights
// reserved. For further information, contact the author, Alan Kaminsky, at
// alan.kaminsky@parallelcrypto.com.
//
// This source file is free software; you can redistribute it and/or modify it
// under the terms of the GNU General Public License as published by the Free
// Software Foundation; either version 3 of the License, or (at your option) any
// later version.
//
// This source file is distributed in the hope that it will be useful, but
// WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
// FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for more
// details.
//
// A copy of the GNU General Public License is provided in the file gpl.txt. You
// may also obtain a copy of the GNU General Public License on the World Wide
// Web at http://www.gnu.org/licenses/gpl.html.
//
//******************************************************************************

#ifndef __GAMMA_CU_INCLUDED__
#define __GAMMA_CU_INCLUDED__

#include "Util.cu"

//------------------------------------------------------------------------------
// This file contains CUDA functions for the gamma function and related
// functions. This file is intended to be #included into a program source file.

static int GAMMA_ITMAX = 200;
static double GAMMA_EPS = 2.22e-16;
static double GAMMA_FPMIN = (2.23e-308 / GAMMA_EPS);

/**
 * Returns the incomplete gamma function P(a,x), evaluated by its series
 * representation. Assumes a > 0 and x >= 0.
 */
static double gser
(double a,
    double x)
{
    double ap, del, sum;
    int i;

    ap = a;
    del = 1.0 / a;
    sum = del;
    for (i = 1; i <= GAMMA_ITMAX; ++i)
    {
        ap += 1.0;
        del *= x / ap;
        sum += del;
        if (fabs(del) < fabs(sum) * GAMMA_EPS)
        {
            return sum * exp(-x + a * log(x) - lgamma(a));
        }
    }
    return 1.0; // Too many iterations
}

/**
 * Returns the complementary incomplete gamma function Q(a,x), evaluated by its
 * continued fraction representation. Assumes a > 0 and x >= 0.
 */
static double gcf
(double a,
    double x)
{
    double b, c, d, h, an, del;
    int i;

    b = x + 1.0 - a;
    c = 1.0 / GAMMA_FPMIN;
    d = 1.0 / b;
    h = d;
    for (i = 1; i <= GAMMA_ITMAX; ++i)
    {
        an = -i * (i - a);
        b += 2.0;
        d = an * d + b;
        if (fabs(d) < GAMMA_FPMIN) d = GAMMA_FPMIN;
        c = b + an / c;
        if (fabs(c) < GAMMA_FPMIN) c = GAMMA_FPMIN;
        d = 1.0 / d;
        del = d * c;
        h *= del;
        if (fabs(del - 1.0) < GAMMA_EPS)
        {
            return exp(-x + a * log(x) - lgamma(a)) * h;
        }
    }
    return 0.0; // Too many iterations
}

/**
 * Returns the incomplete gamma function P(a,x).
 */
static double gammp
(double a,
    double x)
{
    if (a <= 0.0) die("gammp(): a illegal");
    if (x < 0.0) die("gammp(): x illegal");
    return x == 0.0 ? 0.0 : x < a + 1.0 ? gser(a, x) : 1.0 - gcf(a, x);
}

/**
 * Returns the complementary incomplete gamma function Q(a,x) = 1 - P(a,x).
 */
static double gammq
(double a,
    double x)
{
    if (a <= 0.0) die("gammq(): a illegal");
    if (x < 0.0) die("gammq(): x illegal");
    return x == 0.0 ? 1.0 : x < a + 1.0 ? 1.0 - gser(a, x) : gcf(a, x);
}

#endif