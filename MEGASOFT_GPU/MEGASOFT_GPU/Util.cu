#include "hip/hip_runtime.h"
//******************************************************************************
//
// File:    Util.cu
// Author:  Alan Kaminsky
// Version: 19-Jan-2012
//
// This source file is copyright (C) 2012 by Parallel Crypto LLC. All rights
// reserved. For further information, contact the author, Alan Kaminsky, at
// alan.kaminsky@parallelcrypto.com.
//
// This source file is free software; you can redistribute it and/or modify it
// under the terms of the GNU General Public License as published by the Free
// Software Foundation; either version 3 of the License, or (at your option) any
// later version.
//
// This source file is distributed in the hope that it will be useful, but
// WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
// FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for more
// details.
//
// A copy of the GNU General Public License is provided in the file gpl.txt. You
// may also obtain a copy of the GNU General Public License on the World Wide
// Web at http://www.gnu.org/licenses/gpl.html.
//
//******************************************************************************

#ifndef __UTIL_CU_INCLUDED__
#define __UTIL_CU_INCLUDED__

#include <stdlib.h>
#include <stdio.h>
#ifndef _MSC_VER
#include <unistd.h>
#endif
#include<math.h>
#include <sys/types.h>
#include <time.h>
#include <hip/hip_runtime_api.h>

/**
 * Program name (argv[0]).
 */
static char* progname;

/**
 * Print an error message and exit.
 *
 * @param  msg  Error message.
 */
static void die
(char* msg)
{
    fprintf(stderr, "%s: %s\n", progname, msg);
    exit(1);
}

/**
 * If necessary, print a CUDA related error message and exit.
 *
 * @param  err  CUDA error.
 * @param  msg  Error message.
 */
static void checkCuda
(hipError_t err,
    char* msg)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s: %s: %s (%d)\n",
            progname, msg, hipGetErrorString(err), err);
        exit(1);
    }
}

/**
 * Set the CUDA device. The CUDA_DEVICE environment variable specifies the CUDA
 * device to use. If this variable is not set, CUDA device 0 is used. The CUDA
 * device must support compute capability 2.0 or higher.
 */
static void setCudaDevice()
{
    char* CUDA_DEVICE = getenv("CUDA_DEVICE");
    int dev;
    struct hipDeviceProp_t prop;

    if (CUDA_DEVICE == NULL) CUDA_DEVICE = "0";
    if (sscanf(CUDA_DEVICE, "%d", &dev) != 1)
    {
        fprintf(stderr,
            "%s: Environment variable CUDA_DEVICE=\"%s\" invalid\n",
            progname, CUDA_DEVICE);
        exit(1);
    }
    if (hipGetDeviceProperties(&prop, dev) != hipSuccess)
    {
        fprintf(stderr,
            "%s: Could not get properties for CUDA device %d\n",
            progname, dev);
        exit(1);
    }
    if (prop.major < 2 || prop.major == 9999)
    {
        fprintf(stderr,
            "%s: CUDA device %d: %s, compute capability %d.%d, 2.0 required\n",
            progname, dev, prop.name, prop.major, prop.minor);
        exit(1);
    }
    printf("CUDA device %d: %s, compute capability %d.%d\n",
        dev, prop.name, prop.major, prop.minor);
    checkCuda(hipSetDevice(dev), "Could not set CUDA device");
}
#ifndef _MSC_VER
/**
 * Returns the system clock in milliseconds.
 */
static u_int64_t currentTimeMillis()
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec * 1000LLU + tv.tv_usec / 1000LLU;
}
#endif
#endif